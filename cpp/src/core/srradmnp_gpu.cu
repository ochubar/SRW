
#include <hip/hip_runtime.h>
/************************************************************************//**
 * File: srradmnp_gpu.cu
 * Description: Various "manipulations" with Radiation data (e.g. "extraction" of Intensity from Electric Field, etc.) (CUDA implementation)
 * Project: Synchrotron Radiation Workshop
 * First release: 2023
 *
 * Copyright (C) Brookhaven National Laboratory
 * All Rights Reserved
 *
 * @author H.Goel
 * @version 1.0
 ***************************************************************************/

#ifdef _OFFLOAD_GPU
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <assert.h>
#include <math.h>
#include "srradmnp.h"
#include "gmmeth.h"

template <bool allStokesReq, bool intOverEnIsRequired, int PolCom>
__global__ void ExtractSingleElecIntensity2DvsXZ_Kernel(srTRadExtract RadExtract, srTSRWRadStructAccessData RadAccessData, srTRadGenManip *obj, double* arAuxInt, long long ie0, long long ie1, double InvStepRelArg, int Int_or_ReE)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
    int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range
    
	if (ix < RadAccessData.nx && iz < RadAccessData.nz) 
    {
		//int PolCom = RadExtract.PolarizCompon;
			
		//bool allStokesReq = (PolCom == -5); //OC18042020

		float* pI = 0, * pI1 = 0, * pI2 = 0, * pI3 = 0; //OC17042020
		double* pId = 0, * pI1d = 0, * pI2d = 0, * pI3d = 0;
		long ne = RadAccessData.ne, nx = RadAccessData.nx, nz = RadAccessData.nz;
		//float *pI = 0;
		//DOUBLE *pId = 0;
		//double *pId = 0; //OC26112019 (related to SRW port to IGOR XOP8 on Mac)
		long long nxnz = ((long long)nx) * ((long long)nz);
		if (Int_or_ReE != 2)
		{
			pI = RadExtract.pExtractedData;
			if (allStokesReq) //OC17042020
			{
				pI1 = pI + nxnz; pI2 = pI1 + nxnz; pI3 = pI2 + nxnz;
			}
		}
		else
		{
			pId = RadExtract.pExtractedDataD;
			if (allStokesReq) //OC17042020
			{
				pI1d = pId + nxnz; pI2d = pI1d + nxnz; pI3d = pI2d + nxnz;
			}
		}

		float* pEx0 = RadAccessData.pBaseRadX;
		float* pEz0 = RadAccessData.pBaseRadZ;

		//long PerX = RadAccessData.ne << 1;
		//long PerZ = PerX*RadAccessData.nx;
		//long long PerX = RadAccessData.ne << 1;
		//long long PerZ = PerX*RadAccessData.nx;
		long long PerX = ((long long)ne) << 1; //OC18042020
		long long PerZ = PerX * nx;

		//bool intOverEnIsRequired = (RadExtract.Int_or_Phase == 7) && (ne > 1); //OC18042020
		double resInt, resInt1, resInt2, resInt3;
		double ConstPhotEnInteg = 1.;
		long long Two_ie0 = ie0 << 1, Two_ie1 = ie1 << 1; //OC26042019
		long ie;

		long offset = iz * PerZ + ix * PerX;
		long offsetDiv2 = offset >> 1;

		float* pEx_StartForX = pEx0 + offset;
		float* pEz_StartForX = pEz0 + offset;
		if (pI != 0)
		{
			pI += offsetDiv2;
			if (allStokesReq)
			{
				pI1 += offsetDiv2;
				pI2 += offsetDiv2;
				pI3 += offsetDiv2;
			}
		} 

		if (pId != 0)
		{
			pId += offsetDiv2;
			if (allStokesReq)
			{
				pI1d += offsetDiv2;
				pI2d += offsetDiv2;
				pI3d += offsetDiv2;
			}
		} 
		
		//long ixPerX = 0;

		float* pEx_St = pEx_StartForX + Two_ie0;
		float* pEz_St = pEz_StartForX + Two_ie0;
		float* pEx_Fi = pEx_StartForX + Two_ie1;
		float* pEz_Fi = pEz_StartForX + Two_ie1;

		if (intOverEnIsRequired) //OC140813
		{//integrate over photon energy / time
			double* tInt = arAuxInt;
			float* pEx_StAux = pEx_St;
			float* pEz_StAux = pEz_St;

			if (!allStokesReq) //OC17042020
			{
				for (ie = 0; ie < ne; ie++) //OC18042020
				//for(int ie=0; ie<RadAccessData.ne; ie++)
				{
					*(tInt++) = obj->IntensityComponent(pEx_StAux, pEz_StAux, PolCom, Int_or_ReE);
					pEx_StAux += 2;
					pEz_StAux += 2;
				}
				resInt = ConstPhotEnInteg * CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, ne, RadAccessData.eStep); //OC18042020
				//resInt = ConstPhotEnInteg*CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, RadAccessData.ne, RadAccessData.eStep);
			}
			else
			{
				for (ie = 0; ie < ne; ie++)
				{
					*(tInt++) = obj->IntensityComponent(pEx_StAux, pEz_StAux, -1, Int_or_ReE);
					pEx_StAux += 2; pEz_StAux += 2;
				}
				resInt = ConstPhotEnInteg * CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, ne, RadAccessData.eStep);

				tInt = arAuxInt; pEx_StAux = pEx_St; pEz_StAux = pEz_St;
				for (ie = 0; ie < ne; ie++)
				{
					*(tInt++) = obj->IntensityComponent(pEx_StAux, pEz_StAux, -2, Int_or_ReE);
					pEx_StAux += 2; pEz_StAux += 2;
				}
				resInt1 = ConstPhotEnInteg * CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, ne, RadAccessData.eStep);

				tInt = arAuxInt; pEx_StAux = pEx_St; pEz_StAux = pEz_St;
				for (ie = 0; ie < ne; ie++)
				{
					*(tInt++) = obj->IntensityComponent(pEx_StAux, pEz_StAux, -3, Int_or_ReE);
					pEx_StAux += 2; pEz_StAux += 2;
				}
				resInt2 = ConstPhotEnInteg * CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, ne, RadAccessData.eStep);

				tInt = arAuxInt; pEx_StAux = pEx_St; pEz_StAux = pEz_St;
				for (ie = 0; ie < ne; ie++)
				{
					*(tInt++) = obj->IntensityComponent(pEx_StAux, pEz_StAux, -4, Int_or_ReE);
					pEx_StAux += 2; pEz_StAux += 2;
				}
				resInt3 = ConstPhotEnInteg * CGenMathMeth::Integ1D_FuncDefByArray(arAuxInt, ne, RadAccessData.eStep);
			}
		}
		else
		{
			if (!allStokesReq) //OC18042020
			{
				resInt = obj->IntensityComponentSimpleInterpol(pEx_St, pEx_Fi, pEz_St, pEz_Fi, InvStepRelArg, PolCom, Int_or_ReE);
			}
			else //OC18042020
			{
				resInt = obj->IntensityComponentSimpleInterpol(pEx_St, pEx_Fi, pEz_St, pEz_Fi, InvStepRelArg, -1, Int_or_ReE);
				resInt1 = obj->IntensityComponentSimpleInterpol(pEx_St, pEx_Fi, pEz_St, pEz_Fi, InvStepRelArg, -2, Int_or_ReE);
				resInt2 = obj->IntensityComponentSimpleInterpol(pEx_St, pEx_Fi, pEz_St, pEz_Fi, InvStepRelArg, -3, Int_or_ReE);
				resInt3 = obj->IntensityComponentSimpleInterpol(pEx_St, pEx_Fi, pEz_St, pEz_Fi, InvStepRelArg, -4, Int_or_ReE);
			}
		}
		//OC140813
		if (pI != 0) *pI = (float)resInt;
		if (pId != 0) *pId = resInt; //OC18042020
		//if(pId != 0) *(pId++) = (double)resInt;
		if (allStokesReq) //OC18042020
		{
			if (RadExtract.pExtractedData != 0)
			{
				*pI1 = (float)resInt1; *pI2 = (float)resInt2; *pI3 = (float)resInt3;
			}
			else
			{
				*pI1d = resInt1; *pI2d = resInt2; *pI3d = resInt3;
			}
		}
	}
}

template <bool allStokesReq, bool intOverEnIsRequired>
static inline void ExtractSingleElecIntensity2DvsXZ_GPUSub(dim3 &blocks, dim3 &threads, srTRadExtract RadExtract, srTSRWRadStructAccessData RadAccessData, srTRadGenManip *local_copy, double* arAuxInt, long long ie0, long long ie1, double InvStepRelArg, int Int_or_ReE)
{
	switch(RadExtract.PolarizCompon)
	{
		case 5: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 5><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case 4: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 4><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case 3: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 3><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case 2: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 2><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case 1: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 1><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case 0: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, 0><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case -1: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, -1><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case -2: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, -2><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case -3: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, -3><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		case -4: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, -4><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
		default: ExtractSingleElecIntensity2DvsXZ_Kernel<allStokesReq, intOverEnIsRequired, -5><<<blocks, threads>>>(RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE); break;
	}
}

int srTRadGenManip::ExtractSingleElecIntensity2DvsXZ_GPU(srTRadExtract& RadExtract, double* arAuxInt, long long ie0, long long ie1, double InvStepRelArg, TGPUUsageArg* pGPU)
{
	srTSRWRadStructAccessData& RadAccessData = *((srTSRWRadStructAccessData*)(hRadAccessData.ptr()));

    const int bs = 256;
    dim3 blocks(RadAccessData.nx / bs + ((RadAccessData.nx & (bs - 1)) != 0), RadAccessData.nz);
    dim3 threads(bs, 1);

    if (RadAccessData.pBaseRadX != NULL)
	{
		RadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, RadAccessData.pBaseRadX, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
		CAuxGPU::EnsureDeviceMemoryReady(pGPU, RadAccessData.pBaseRadX);
	}
	if (RadAccessData.pBaseRadZ != NULL)
	{
		RadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, RadAccessData.pBaseRadZ, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
		CAuxGPU::EnsureDeviceMemoryReady(pGPU, RadAccessData.pBaseRadZ);
	}

	srTRadGenManip *local_copy = (srTRadGenManip*)CAuxGPU::ToDevice(pGPU, this, sizeof(srTRadGenManip));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, local_copy);

    arAuxInt = (double*)CAuxGPU::ToDevice(pGPU, arAuxInt, RadAccessData.ne*sizeof(double));
    CAuxGPU::EnsureDeviceMemoryReady(pGPU, arAuxInt);

	bool allStokesReq = (RadExtract.PolarizCompon == -5);
	bool intOverEnIsRequired = (RadExtract.Int_or_Phase == 7) && (RadAccessData.ne > 1);

	int Int_or_ReE = RadExtract.Int_or_Phase;
	if (Int_or_ReE == 7) Int_or_ReE = 0; //OC150813: time/phot. energy integrated single-e intensity requires "normal" intensity here

	if (allStokesReq)
		if (intOverEnIsRequired)
			ExtractSingleElecIntensity2DvsXZ_GPUSub<true, true> (blocks, threads, RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE);
		else
			ExtractSingleElecIntensity2DvsXZ_GPUSub<true, false> (blocks, threads, RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE);
	else
		if (intOverEnIsRequired)
			ExtractSingleElecIntensity2DvsXZ_GPUSub<false, true> (blocks, threads, RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE);
		else
			ExtractSingleElecIntensity2DvsXZ_GPUSub<false, false> (blocks, threads, RadExtract, RadAccessData, local_copy, arAuxInt, ie0, ie1, InvStepRelArg, Int_or_ReE);
	
    CAuxGPU::ToHostAndFree(pGPU, local_copy, sizeof(srTRadGenManip), true);
    CAuxGPU::ToHostAndFree(pGPU, arAuxInt, RadAccessData.ne*sizeof(double), true);
	CAuxGPU::MarkUpdated(pGPU, RadAccessData.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, RadAccessData.pBaseRadZ, true, false);

#ifndef _DEBUG
	if (RadAccessData.pBaseRadX != NULL)
		RadAccessData.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, RadAccessData.pBaseRadX);
	if (RadAccessData.pBaseRadZ != NULL)
		RadAccessData.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, RadAccessData.pBaseRadZ);
#endif

#ifdef _DEBUG
	if (RadAccessData.pBaseRadX != NULL)
		RadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, RadAccessData.pBaseRadX, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
	if (RadAccessData.pBaseRadZ != NULL)
		RadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, RadAccessData.pBaseRadZ, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
	cudaStreamSynchronize(0);
	auto err = cudaGetLastError();
	printf("%s\r\n", cudaGetErrorString(err));
#endif
	return 0;
}

template <int PolCom, bool EhOK, bool EvOK, int gt1_iter, int itPerBlk>
__global__ void ExtractSingleElecMutualIntensityVsXZ_Kernel(const float* __restrict__ pEx0, const float* __restrict__ pEz0, float* __restrict__ pMI0, long nxnz, long itStart, long itEnd, long PerX, long iter0)
{
	//Calculate coordinates as the typical triangular matrix
	int i0 = (blockIdx.x * blockDim.x + threadIdx.x); //<=nxnz range
	int it0_0 = (blockIdx.y * blockDim.y + threadIdx.y); //nxnz/(2*itPerBlk) range
	long iter = iter0;

	if (i0 > nxnz) return;
	if (it0_0 > nxnz / 2) return;

	for (int it0 = it0_0 * itPerBlk; it0 < it0_0 * itPerBlk + itPerBlk; it0++)
	{
		long it = it0;
		long i = i0;
		if (i0 > it0) //If the coordinates are past the triangular bounds, switch to the lower half of the triangle
		{
			it = nxnz - it0 - 1;
			i = i0 - (it0 + 1);
		}

		if (it >= itEnd) {
			return;
		}

		//float* pMI = pMI0 + it0 * (nxnz << 1) + (i0 << 1); //Compact representation coordinates
		float* pMI = pMI0 + (it - itStart) * (nxnz << 1) + (i << 1); //Full representation coordinates
		const float* pEx = pEx0 + i * PerX;
		const float* pEz = pEz0 + i * PerX;
		const float* pExT = pEx0 + (it - itStart) * PerX;
		const float* pEzT = pEz0 + (it - itStart) * PerX;

		float ExRe = 0., ExIm = 0., EzRe = 0., EzIm = 0.;
		float ExReT = 0., ExImT = 0., EzReT = 0., EzImT = 0.;

		{
			if (EhOK)
			{
				ExRe = *pEx; ExIm = *(pEx + 1);
				if (i != (it - itStart)) {
					ExReT = *pExT; ExImT = *(pExT + 1);
				}
				else {
					ExReT = ExRe;
					ExImT = ExIm;
				}
			}
			if (EvOK) {
				EzRe = *pEz; EzIm = *(pEz + 1);
				if (i != (it - itStart)) {
					EzReT = *pEzT; EzImT = *(pEzT + 1);
				}
				else {
					EzReT = EzRe;
					EzImT = EzIm;
				}
			}
		}
		float ReMI = 0., ImMI = 0.;

		switch (PolCom)
		{
		case 0: // Lin. Hor.
		{
			ReMI = ExRe * ExReT + ExIm * ExImT;
			ImMI = ExIm * ExReT - ExRe * ExImT;
			break;
		}
		case 1: // Lin. Vert.
		{
			ReMI = EzRe * EzReT + EzIm * EzImT;
			ImMI = EzIm * EzReT - EzRe * EzImT;
			break;
		}
		case 2: // Linear 45 deg.
		{
			float ExRe_p_EzRe = ExRe + EzRe, ExIm_p_EzIm = ExIm + EzIm;
			float ExRe_p_EzReT = ExReT + EzReT, ExIm_p_EzImT = ExImT + EzImT;
			ReMI = 0.5f * (ExRe_p_EzRe * ExRe_p_EzReT + ExIm_p_EzIm * ExIm_p_EzImT);
			ImMI = 0.5f * (ExIm_p_EzIm * ExRe_p_EzReT - ExRe_p_EzRe * ExIm_p_EzImT);
			break;
		}
		case 3: // Linear 135 deg.
		{
			float ExRe_mi_EzRe = ExRe - EzRe, ExIm_mi_EzIm = ExIm - EzIm;
			float ExRe_mi_EzReT = ExReT - EzReT, ExIm_mi_EzImT = ExImT - EzImT;
			ReMI = 0.5f * (ExRe_mi_EzRe * ExRe_mi_EzReT + ExIm_mi_EzIm * ExIm_mi_EzImT);
			ImMI = 0.5f * (ExIm_mi_EzIm * ExRe_mi_EzReT - ExRe_mi_EzRe * ExIm_mi_EzImT);
			break;
		}
		case 5: // Circ. Left //OC08092019: corrected to be in compliance with definitions for right-hand frame (x,z,s) and with corresponding definition and calculation of Stokes params
			//case 4: // Circ. Right
		{
			float ExRe_mi_EzIm = ExRe - EzIm, ExIm_p_EzRe = ExIm + EzRe;
			float ExRe_mi_EzImT = ExReT - EzImT, ExIm_p_EzReT = ExImT + EzReT;
			ReMI = 0.5f * (ExRe_mi_EzIm * ExRe_mi_EzImT + ExIm_p_EzRe * ExIm_p_EzReT);
			ImMI = 0.5f * (ExIm_p_EzRe * ExRe_mi_EzImT - ExRe_mi_EzIm * ExIm_p_EzReT);
			break;
		}
		case 4: // Circ. Right //OC08092019: corrected to be in compliance with definitions for right-hand frame (x,z,s) and with corresponding definition and calculation of Stokes params
			//case 5: // Circ. Left
		{
			float ExRe_p_EzIm = ExRe + EzIm, ExIm_mi_EzRe = ExIm - EzRe;
			float ExRe_p_EzImT = ExReT + EzImT, ExIm_mi_EzReT = ExImT - EzReT;
			ReMI = 0.5f * (ExRe_p_EzIm * ExRe_p_EzImT + ExIm_mi_EzRe * ExIm_mi_EzReT);
			ImMI = 0.5f * (ExIm_mi_EzRe * ExRe_p_EzImT - ExRe_p_EzIm * ExIm_mi_EzReT);
			break;
		}
		case -1: // s0
		{
			ReMI = ExRe * ExReT + ExIm * ExImT + EzRe * EzReT + EzIm * EzImT;
			ImMI = ExIm * ExReT - ExRe * ExImT + EzIm * EzReT - EzRe * EzImT;
			break;
		}
		case -2: // s1
		{
			ReMI = ExRe * ExReT + ExIm * ExImT - (EzRe * EzReT + EzIm * EzImT);
			ImMI = ExIm * ExReT - ExRe * ExImT - (EzIm * EzReT - EzRe * EzImT);
			break;
		}
		case -3: // s2
		{
			ReMI = ExImT * EzIm + ExIm * EzImT + ExReT * EzRe + ExRe * EzReT;
			ImMI = ExReT * EzIm - ExRe * EzImT - ExImT * EzRe + ExIm * EzReT;
			break;
		}
		case -4: // s3
		{
			ReMI = ExReT * EzIm + ExRe * EzImT - ExImT * EzRe - ExIm * EzReT;
			ImMI = ExIm * EzImT - ExImT * EzIm - ExReT * EzRe + ExRe * EzReT;
			break;
		}
		default: // total mutual intensity, same as s0
		{
			ReMI = ExRe * ExReT + ExIm * ExImT + EzRe * EzReT + EzIm * EzImT;
			ImMI = ExIm * ExReT - ExRe * ExImT + EzIm * EzReT - EzRe * EzImT;
			break;
			//return CAN_NOT_EXTRACT_MUT_INT;
		}
		}

		if (gt1_iter > 0)
		{
			pMI[0] = (pMI[0] * iter + (float)ReMI) / (float)(iter + 1.);
			pMI[1] = (pMI[1] * iter + (float)ImMI) / (float)(iter + 1.);
		}
		else if (gt1_iter == 0)
		{
			pMI[0] = (float)ReMI;
			pMI[1] = (float)ImMI;
		}
		else
		{
			pMI[0] += (float)ReMI;
			pMI[1] += (float)ImMI;
		}
	}
}

template <int PolCom, int gt1_iter>
int ExtractSingleElecMutualIntensityVsXZ_GPUSub(float* pEx, float* pEz, float* pMI0, long nx, long nz, long ne, long itStart, long itEnd, long PerX, long iter, bool EhOK, bool EvOK, TGPUUsageArg* pGPU)
{
	long long nxnz = ((long long)nx) * ((long long)nz);

	const int itPerBlk = 1;
	dim3 threads = dim3(48, 16, 1);
	dim3 grid = dim3((nxnz + 1) / threads.x + (threads.x > 1), (nxnz / 2) / (threads.y * itPerBlk) + (threads.y > 1), 1);

	pEx = (float*)CAuxGPU::ToDevice(pGPU, pEx, nxnz*2*sizeof(float));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pEx);

	pEz = (float*)CAuxGPU::ToDevice(pGPU, pEz, nxnz*2*sizeof(float));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pEz);

	pMI0 = (float*)CAuxGPU::ToDevice(pGPU, pMI0, (itEnd - itStart)*nxnz*2*sizeof(float));

	if (EhOK)
	{
		if (EvOK) ExtractSingleElecMutualIntensityVsXZ_Kernel<PolCom, true, true, gt1_iter, itPerBlk> << <grid, threads >> > (pEx, pEz, pMI0, nxnz, itStart, itEnd, PerX, iter);
		else ExtractSingleElecMutualIntensityVsXZ_Kernel<PolCom, true, false, gt1_iter, itPerBlk> << <grid, threads >> > (pEx, pEz, pMI0, nxnz, itStart, itEnd, PerX, iter);
	}
	else
	{
		if (EvOK) ExtractSingleElecMutualIntensityVsXZ_Kernel<PolCom, false, true, gt1_iter, itPerBlk> << <grid, threads >> > (pEx, pEz, pMI0, nxnz, itStart, itEnd, PerX, iter);
		else ExtractSingleElecMutualIntensityVsXZ_Kernel<PolCom, false, false, gt1_iter, itPerBlk> << <grid, threads >> > (pEx, pEz, pMI0, nxnz, itStart, itEnd, PerX, iter);
	}

	pEx = (float*)CAuxGPU::ToHostAndFree(pGPU, pEx, nxnz * 2 * sizeof(float), true);
	pEz = (float*)CAuxGPU::ToHostAndFree(pGPU, pEz, nxnz * 2 * sizeof(float), true);
	
	CAuxGPU::MarkUpdated(pGPU, pMI0, true, false);

#ifdef _DEBUG
	if (pMI0 != NULL)
		pMI0 = (float*)CAuxGPU::ToHostAndFree(pGPU, pMI0, (itEnd - itStart)*ne*nx*nz*2*sizeof(float));

	cudaStreamSynchronize(0);
	auto err = cudaGetLastError();
	printf("%s\r\n", cudaGetErrorString(err));
#endif
	return 0;
}

int srTRadGenManip::ExtractSingleElecMutualIntensityVsXZ_GPU(float* pEx, float* pEz, float* pMI0, long nx, long nz, long ne, long itStart, long itEnd, long PerX, long iter, int PolCom, bool EhOK, bool EvOK, TGPUUsageArg* pGPU)
{
	if (iter > 0)
	{
		switch (PolCom)
		{
		case  1: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  1, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  2: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  2, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  0: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  0, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  3: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  3, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  4: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  4, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  5: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  5, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -1: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -1, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -2: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -2, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -3: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -3, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -4: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -4, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		default: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -5, 1>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		}
	}
	else if (iter == 0)
	{
		switch (PolCom)
		{
		case  0: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  0, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  1: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  1, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  2: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  2, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  3: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  3, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  4: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  4, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case  5: return ExtractSingleElecMutualIntensityVsXZ_GPUSub<  5, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -1: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -1, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -2: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -2, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -3: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -3, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		case -4: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -4, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		default: return ExtractSingleElecMutualIntensityVsXZ_GPUSub< -5, 0>(pEx, pEz, pMI0, nx, nz, ne, itStart, itEnd, PerX, iter, EhOK, EvOK, pGPU);
		}
	}
}

#endif